#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

__global__ void Transpose(int *d_a,int r,int c){

int i = blockIdx.x*blockDim.x+threadIdx.x;
int j = blockIdx.y*blockDim.y+threadIdx.y;

__syncthreads();

if(i<c && j<r)
{
    int id1 = i+j*c;
    int id2 = j+i*r;
	int t = d_a[id1];
	__syncthreads();
	d_a[id2]=t;
	
}
	
}

int main()
{
	int r,c,i,j;
	cout<<"Enter the number of rows and columns:\n";
	cin>>r>>c;
	int h_a[r][c]={0},h_b[c][r];
	for(i=0;i<r;i++)
	{
		for(j=0;j<c;j++)
		h_a[i][j]=2*i+j;
	}
	cout<<"Given array is:\n";
	for(i=0;i<r;i++)
	{
		for(j=0;j<c;j++)
		cout<<h_a[i][j]<<" ";
		cout<<"\n";
	}
	int *d_a;
	hipMalloc((void**)&d_a, r*c*sizeof(int));

	hipMemcpy(d_a, h_a, r*c*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil(1.0*c/dimBlock.x), (int)ceil(1.0*r/dimBlock.y));
	Transpose<<<dimGrid,dimBlock>>>(d_a,r,c);
	hipMemcpy(h_b, d_a, r*c*sizeof(int), hipMemcpyDeviceToHost);
	cout<<"The transpose matrix is:\n";
	for(i=0;i<c;i++)
	{
		for(j=0;j<r;j++)
		cout<<h_b[i][j]<<" ";
		cout<<"\n";
	}

	hipFree(d_a);
	return 0;
}
