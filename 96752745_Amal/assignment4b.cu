
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Sum(int *d_out,int *d_a,int arraySize){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	__shared__ int s_a[1024];

     if(id < arraySize)
        s_a[tid] = d_a[id];
    __syncthreads();
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=arraySize || id+s>=arraySize)
            continue;
        if(tid<s)
            s_a[tid] += s_a[tid+s];
    }
    __syncthreads();
    if(tid==0)
        d_out[bid] = s_a[tid]; 
}

__global__ void Prod(int *d_a,int *d_b,int *d_p,int arraySize){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < arraySize)
	d_p[id] = d_a[id]*d_b[id];	
}

int main()
{
	int arraySize;
	cout<<"Enter array size\n";
	cin>>arraySize;
	int h_a[arraySize],h_b[arraySize],i,h_sum;

	for(i=0;i<arraySize;i++)
	{
		h_a[i]=2*i;
		h_b[i]=3*i;
	}
	int *d_a,*d_b,*d_out,*d_sum,*d_p;
	hipMalloc((void**)&d_a, sizeof(int)*arraySize);
	hipMalloc((void**)&d_b, sizeof(int)*arraySize);
	hipMalloc((void**)&d_p, sizeof(int)*arraySize);
	hipMalloc((void**)&d_out, ceil(1.0*arraySize/1024)*sizeof(int));
	hipMalloc((void**)&d_sum,sizeof(int));

	hipMemcpy(d_a,h_a,arraySize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,arraySize*sizeof(int),hipMemcpyHostToDevice);
	Prod<<<ceil(1.0*arraySize/1024),1024>>>(d_a,d_b,d_p,arraySize);

	Sum<<<ceil(arraySize*1.0/1024), 1024>>> (d_out, d_p, arraySize);
    Sum<<<1, 1024>>> (d_sum, d_out, ceil(arraySize*1.0/1024));
    hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    cout<<"Dot Product is "<<h_sum<<endl;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_p);
    hipFree(d_out);
    hipFree(d_sum);
}
