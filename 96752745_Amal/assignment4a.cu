
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Max(int *d_out,int *d_a,int arraySize){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ int s_a[1024];

     if(id < arraySize)
        s_a[tid] = d_a[id];
    __syncthreads();
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=arraySize || id+s>=arraySize)
            continue;
        if(tid<s)
            s_a[tid] = s_a[tid]>s_a[tid + s]?s_a[tid]:s_a[tid+s];
    }
    __syncthreads();
    if(tid==0)
        d_out[bid] = s_a[tid]; 
}

__global__ void Min(int *d_out,int *d_a,int arraySize){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ int s_a[1024];

     if(id < arraySize)
        s_a[tid] = d_a[id];
    __syncthreads();
    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=arraySize || id+s>=arraySize)
            continue;
        if(tid<s)
            s_a[tid] = s_a[tid]<s_a[tid + s]?s_a[tid]:s_a[tid+s];
    }
    __syncthreads();
    if(tid==0)
        d_out[bid] = s_a[tid]; 
}

int main()
{
	int arraySize,max=0,min;
	cout<<"Enter array size\n";
	cin>>arraySize;
	int h_a[arraySize],i,h_max,h_min;
	for(i=0;i<arraySize;i++)
	h_a[i]=5*i;
	min=h_a[0];
	for(i=0;i<arraySize;i++)
	{
		if(h_a[i]>max)
		max=h_a[i];
		if(h_a[i]<min)
		min=h_a[i];
	}

	int *d_a,*d_out1,*d_out2,*d_max,*d_min;
	hipMalloc((void**)&d_a,arraySize*sizeof(int));
	hipMalloc((void**)&d_out1, ceil(arraySize*1.0/1024)*sizeof(int));
	hipMalloc((void**)&d_out2, ceil(arraySize*1.0/1024)*sizeof(int));
    hipMalloc((void**)&d_max, sizeof(int));
    hipMalloc((void**)&d_min, sizeof(int));

    hipMemcpy(d_a,h_a,arraySize*sizeof(int),hipMemcpyHostToDevice);

    Max<<<ceil(arraySize*1.0/1024), 1024>>> (d_out1, d_a, arraySize);
    Max<<<1, 1024>>> (d_max, d_out1, ceil(arraySize*1.0/1024));
    hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
    
    Min<<<ceil(arraySize*1.0/1024), 1024>>> (d_out2, d_a, arraySize);
    Min<<<1, 1024>>> (d_min, d_out2, ceil(arraySize*1.0/1024));
    hipMemcpy(&h_min, d_min, sizeof(int), hipMemcpyDeviceToHost);

    if(h_max==max)
    cout<<"Max element is "<<h_max<<endl;
    else
    cout<<"Some error has occured for calculating max!!"<<endl;

    if(h_min==min)
    cout<<"Min element is "<<h_min<<endl;
    else
    cout<<"Some error has occured for calculating min!!"<<endl;

    hipFree(d_a);
    hipFree(d_out1);
    hipFree(d_out2);
    hipFree(d_max);
    hipFree(d_min);
}
